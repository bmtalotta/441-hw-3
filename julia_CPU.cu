#include "FreeImage.h"
#include "stdio.h"

#define DIM 2000

struct hipComplex{
        float   r;
        float   i;
        hipComplex( float a, float b) : r(a), i(b) {};
        float magnitude2(void) { return r*r + i*i; }
        hipComplex operator*(const hipComplex& a) {
                return hipComplex( r * a.r - i * a.i, i * a.r + r * a.i );
        }
        hipComplex operator+(const hipComplex& a) {
                return hipComplex(r+ a.r, i + a.i);
        }
};



int julia(int x, int y)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);
    int i = 0;
    for (i = 0; i < 200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000) return 0;
    }
    return 1;
}


void kernel(char* ptr)
{
    for (int y = 0; y < DIM; y++)
        for (int x = 0; x < DIM; x++)
        {
            int offset = x + y * DIM;
            ptr[offset] = julia(x, y);
        }
}


int main()
{
    FreeImage_Initialise();
    atexit(FreeImage_DeInitialise);
    FIBITMAP* bitmap = FreeImage_Allocate(DIM, DIM, 24);

    char charmap[DIM][DIM];
    kernel(&charmap[0][0]);

    RGBQUAD color;
    for (int i = 0; i < DIM; i++) {
        for (int j = 0; j < DIM; j++) {
            color.rgbRed = 0;
            color.rgbGreen = 0;
            color.rgbBlue = 0;
            if (charmap[i][j] == 1)
                color.rgbGreen = 255.0;
            FreeImage_SetPixelColor(bitmap, i, j, &color);
        }
    }
    FreeImage_Save(FIF_PNG, bitmap, "output.png", 0);
    FreeImage_Unload(bitmap);

    return 0;
}
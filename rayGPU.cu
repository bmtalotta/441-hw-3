#include "hip/hip_runtime.h"
// This is a simple ray tracer that shoots rays top down toward randomly
// generates spheres and draws the sphere in a random color based on where
// the ray hits it.
//modified by Ben Talotta
//translated to gpu by reference of julia_gpu.cu
#include "FreeImage.h"
#include "stdio.h"

#define DIM 2048
#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f

struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
    // Tells us if a ray hits the sphere; return the
    // depth of the hit, or -infinity if the ray misses the sphere
    __device__ float hit( float ox, float oy, float *n ) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius)
        {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};

#define SPHERES 80

// Loops through each pixel in the image (represented by arrays of
// red, green, and blue) and then for each pixel checks if a ray from
// top down hits one of the randomly generated spheres.
// If so, calculate a shade of color based on where the ray hits it.
 __global__ void drawSpheres(Sphere spheres[], char *red, char *green, char *blue)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * DIM;
	float   ox = (x - DIM/2);
	float   oy = (y - DIM/2);

	float   r=0, g=0, b=0;
	float   maxz = -INF;
	for(int i=0; i<SPHERES; i++)
 	{
        float   n;
        float   t = spheres[i].hit( ox, oy, &n );
        if (t > maxz){
			// Scale RGB color based on z depth of sphere
        	float fscale = n;
        	r = spheres[i].r * fscale;
        	g = spheres[i].g * fscale;
        	b = spheres[i].b * fscale;
        	maxz = t;
        }
     }
    int offset = x + y * DIM;
    red[offset] = (char) (r * 255);
    green[offset] = (char) (g * 255);
    blue[offset] = (char) (b * 255);

}

int main()
{
    FreeImage_Initialise();
    atexit(FreeImage_DeInitialise);
    FIBITMAP * bitmap = FreeImage_Allocate(DIM, DIM, 24);
    srand(time(NULL));

    char *red;
    char *green;
    char *blue;
    char *dev_r, *dev_g, *dev_b;
    dim3 grid(DIM, DIM);
  // Dynamically create enough memory for DIM * DIM array of char.
  // By making these dynamic rather than auto (e.g. char red[DIM][DIM])
  // we can make them much bigger since they are allocated off the heap
  red = (char *) malloc(DIM*DIM*sizeof(char));
  green = (char *) malloc(DIM*DIM*sizeof(char));
  blue = (char *) malloc(DIM*DIM*sizeof(char));
  hipMalloc((void**)&dev_r, DIM * DIM* sizeof(char));
  hipMalloc((void**)&dev_g, DIM * DIM* sizeof(char));
  hipMalloc((void**)&dev_b, DIM * DIM* sizeof(char));

  // Create random spheres at different coordinates, colors, radius
  Sphere spheres[SPHERES];
  Sphere *dev_s;
  for (int i = 0; i<SPHERES; i++)
  {
    spheres[i].r = rnd( 1.0f );
    spheres[i].g = rnd( 1.0f );
    spheres[i].b = rnd( 1.0f );
    spheres[i].x = rnd( (float) DIM ) - (DIM/2.0);
    spheres[i].y = rnd( (float) DIM ) - (DIM/2.0);
    spheres[i].z = rnd( (float) DIM ) - (DIM/2.0);
    spheres[i].radius = rnd( 200.0f ) + 40;
  }
  
  hipMalloc((void**)&dev_s, SPHERES* sizeof(Sphere));
  hipMemcpy(dev_s, spheres, SPHERES* sizeof(Sphere), hipMemcpyHostToDevice);
  drawSpheres<<<grid, 1>>>(dev_s, dev_r, dev_g, dev_b);
  hipMemcpy(red, dev_r, DIM * DIM * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(green, dev_g, DIM * DIM * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(blue, dev_b, DIM * DIM * sizeof(char), hipMemcpyHostToDevice);

  RGBQUAD color;
  for (int i = 0; i < DIM; i++)
  {
    for (int j = 0; j < DIM; j++)
    {
      int index = j*DIM + i;
      color.rgbRed = red[index];
      color.rgbGreen = green[index];
      color.rgbBlue = blue[index];
      FreeImage_SetPixelColor(bitmap, i, j, &color);
    }
  }
	
  FreeImage_Save(FIF_PNG, bitmap, "ray.png", 0);
  FreeImage_Unload(bitmap);
  free(red);
  free(green);
  free(blue);
    hipFree(dev_r);
    hipFree(dev_g);
    hipFree(dev_b);
    hipFree(dev_s);
  return 0;
}


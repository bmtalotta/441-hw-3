#include "hip/hip_runtime.h"
#include "stdio.h"
#include <stdlib.h>
//based on cuda summing_Arrrays example
#define N 8 * 1000000

__global__ void findMin(int* a, int low, int high, int minVal)
{
    for(int i = low; i < high; i++ ){
        if(minVal > a[i]){
            minVal = a[i];
        }
    }
}

int main()
{
    dim3 grid(1);
    dim3 threads(8);
    int a[N];
    int *dev_a;
    hipMalloc((void**)&dev_a, 8 * sizeof(int));
    //fill array
    for (int i = 0; i < N; i++){
        a[i] = rand() % 1000000000;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    int numToSort = N / 8;
    int low = numToSort * threadIdx.x;
    int high = low + numToSort - 1;
    int MinVal[8];
    for(int i = 0; i < 8; i++){
        minVal[i] = 1000000000;
    }
    findMin <<<grid, threads >>> (a,low,high,minVal[threadIdx.x]);
    int minAnswer = MinVal[0];
    for(int i = 0; i < 8; i++){
        if(minAnswer > MinVal[i]){
            minAnswer = minVal[i];
        }
    }
    printf("minimum value using cuda is: %d\n", min);

    return 0;
}


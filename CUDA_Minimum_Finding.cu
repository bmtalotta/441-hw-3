#include "hip/hip_runtime.h"
#include "stdio.h"
#include <stdlib.h>
//based on cuda summing_Arrrays example
#define N 8 * 1000000
#define MINE 8
__global__ void findMin(int* a, int* c )
{
    int numToSort = N / 8;
    int low = numToSort * threadIdx.x;
    int high = low + numToSort - 1;
    int minValForThread = 1000000000;
    
    for(int i = low; i < high; i++ ){
        if(minValForThread > a[i]){
            minValForThread = a[i];
        }
    }
    c[threadIdx.x] = minValForThread;
}

int main()
{
    dim3 grid(1);
    dim3 threads(8);
    int a[N];
    int *dev_a;
    int c[8];
    int *dev_c;
    
    for(int i = 0; i < 8; i++){
        c[i] = 1000000000;
    }
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_c, 8 * sizeof(int));
    //fill array
    for (int i = 0; i < N; i++){
        a[i] = rand() % 1000000000;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    findMin <<<grid, threads >>> (a,low,high,minVal[threadIdx.x]);
    
    hipMemcpy(c, dev_c, MINE * sizeof(int), hipMemcpyDeviceToHost);
    int min = c[0];
    for(int i = 0; i < 8; i++){
        if(min> c[i]){
            min= c[i];
        }
    }
    printf("minimum value using cuda is: %d\n", min);
    hipFree(dev_a);
    return 0;
}



#include <hip/hip_runtime.h>
#include "stdio.h"
#include <stdlib.h>
//based on cuda summing_Arrrays example
#define N 8 * 1000000
#define ThreadCount 8
__global__ void findMin(int* a, int* c )
{
    int numToSort = N / 8;
    int low = numToSort * threadIdx.x;
    int high = low + numToSort - 1;
    int minValForThread = 1000000000;
    
    for(int i = low; i < high; i++ ){
        
        //printf("here\n");
        if(minValForThread > a[i]){
            minValForThread = a[i];
        }
    }
    printf("here 3\n");
    
    printf("min for thread %d: %d\n", threadIdx.x, c[threadIdx.x]);
    c[threadIdx.x] = minValForThread;
}

int main()
{
    dim3 grid(1000000000);
    dim3 threads(ThreadCount);
    int *a;
    a = (int *)malloc(sizeof(int) * N);
    int *dev_a;
    int c[8];
    int *dev_c;
    
    for(int i = 0; i < 8; i++){
        c[i] = 1000000000;
    }
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_c, ThreadCount * sizeof(int));
    //fill array
    for (int i = 0; i < N; i++){
        a[i] = rand() % 1000000000;
    }
    
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, ThreadCount * sizeof(int), hipMemcpyHostToDevice);
    findMin <<<grid, threads >>> (a, c);
    hipDeviceSynchronize();
    
    hipMemcpy(c, dev_c, ThreadCount * sizeof(int), hipMemcpyDeviceToHost);
    int min = c[0];
    for(int i = 0; i < 8; i++){
        if(min > c[i]){
            min = c[i];
        }
        
    //   printf("min for thread %d: %d\n", i, c[i]);
    }
    printf("minimum value using cuda is: %d\n", min);
    hipFree(dev_a);
    hipFree(dev_c);
    return 0;
}


//CUDE_Minimum_Fineding.cu
//Ben Talotta

#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
//based on cuda summing_Arrrays example
#define N 8000000
#define ThreadCount 8
__global__ void findMin(int* a, int* c )
{

    int numToSort = N / 8;
    int low = numToSort * threadIdx.x;
    int high = low + numToSort - 1;
    int minValForThread = a[low];
    for(int i = low; i < high; ++i){
        if(minValForThread > a[i]){
            minValForThread = a[i];
        }
    }
    c[threadIdx.x] = minValForThread;
}

int main()
{
    dim3 grid(1);
    int *a;
    a = (int *)malloc(sizeof(int) * N);
    int *dev_a;
    int c[8];
    int *dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_c, ThreadCount * sizeof(int));

    for(int i = 0; i < 8; i++){
        c[i] = 1000000000;
    }
    
    //fill array
    for (int i = 0; i < N; i++){
        a[i] = rand() % 1000000000;
    }
    
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, ThreadCount * sizeof(int), hipMemcpyHostToDevice);
    findMin <<<grid, ThreadCount >>> (dev_a, dev_c);

    hipMemcpy(c, dev_c, ThreadCount * sizeof(int), hipMemcpyDeviceToHost);
    int min = c[0];
    for(int i = 0; i < 8; i++){
        if(min > c[i]){
            min = c[i];
        }
    }
    printf("minimum value using cuda is: %d\n", min);
    hipFree(dev_a);
    hipFree(dev_c);
    return 0;
}

 
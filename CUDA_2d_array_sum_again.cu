//CUDE_2d_arraySum_again.cu
//Ben Talotta

#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 8
#define ROWS 8
//based off sum2darr.cu and kernal test code examples code
__global__ void add(int* a,int* c)
{
    __shared__ int cache[COLUMNS];
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    int x = threadIdx.x;
    cache[x] = a[tid];
    int calculationInBox = blockDim.x / 2;
    while (calculationInBox >= 1)
    {
        if(x < calculationInBox){
            cache[x] += cache[x + calculationInBox];
            __syncthreads();
            if(calculationInBox == 1){
                break;
            }
        }
        calculationInBox /=2;
    }
    if(x == 0){
        c[blockIdx.x] = cache[0];
    }
}

int main()
{
    int a[ROWS][COLUMNS];
    int c[ROWS];
    int* dev_a;
    int* dev_c;
    dim3 grid(ROWS);
    dim3 threads(COLUMNS);


    hipMalloc((void**)&dev_a, ROWS * COLUMNS * sizeof(int));
    hipMalloc((void**)&dev_c, ROWS * sizeof(int));

    for (int y = 0; y < ROWS; y++)              // Fill Arrays
        for (int x = 0; x < COLUMNS; x++)
            a[y][x] = rand()% 10;

    hipMemcpy(dev_a, a, ROWS * COLUMNS * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, ROWS * sizeof(int), hipMemcpyHostToDevice);
    add <<<grid, threads >>> (dev_a, dev_c);
    hipMemcpy(c, dev_c, ROWS * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();//wait for threads to finish
    int findColSum = 0;
    for(int i = 0; i < COLUMNS; i++){
        printf("+ %d",c[i]);
        findColSum += c[i];
    }
    printf("\nthe sum of the columns is: %d\n", findColSum);
    hipFree(dev_a);
    hipFree(dev_c);
    return 0;
}


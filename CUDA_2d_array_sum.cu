
#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 4
#define ROWS 3
//based off sum2darr.cu example code
__global__ void add(int* a,int* c)
{
    int x = threadIdx.x;
    int sum = 0;
    for(int i = 0; i < ROWS; i ++){
        sum += a[(COLUMNS * i) + x];
    }
    
    printf("the sum of the %d thread column is: %d\n", x, sum);
    c[x] = sum;
}

int main()
{
    int a[ROWS][COLUMNS];
    int c[COLUMNS];
    int* dev_a;
    int* dev_c;


    hipMalloc((void**)&dev_a, ROWS * COLUMNS * sizeof(int));
    hipMalloc((void**)&dev_c, COLUMNS * sizeof(int));

    for (int y = 0; y < ROWS; y++)              // Fill Arrays
        for (int x = 0; x < COLUMNS; x++)
            a[y][x] = rand()% 10;

    hipMemcpy(dev_a, a, ROWS * COLUMNS * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, COLUMNS * sizeof(int), hipMemcpyHostToDevice);
    add <<<1, COLUMNS >>> (dev_a, dev_c);
    hipMemcpy(c, dev_c, COLUMNS * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();//wait for threads to finish
    int findColSum = 0;
    for(int i = 0; i < COLUMNS; i++){
        
        printf("the sum of the columns is: %d\n", findColSum);
        findColSum += c[i];
    }
    printf("the sum of the columns is: %d\n", findColSum);
    hipFree(dev_a);
    hipFree(dev_c);
    return 0;
}


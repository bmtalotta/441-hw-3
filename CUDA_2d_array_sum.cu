#include "hip/hip_runtime.h"
#include "stdio.h"
#define COLUMNS 4
#define ROWS 3
//based off sum2darr.cu example code
__global__ void add(int* a,int* c)
{
    int x = blockIdx.x;
    int sum = 0;
    for(int i = 0; i < rows; i ++){
        sum += a[(COLUMNS * i) + x];
    }
    c[x] = sum;
}

int main()
{
    int a[ROWS][COLUMNS];
    int c[COLUMNS];
    int* dev_a;
    int* dev_c;


    hipMalloc((void**)&dev_a, ROWS * COLUMNS * sizeof(int));
    hipMalloc((void**)&dev_c, COLUMNS * sizeof(int));

    for (int y = 0; y < ROWS; y++)              // Fill Arrays
        for (int x = 0; x < COLUMNS; x++)
            a[y][x] = rand()% 50;

    hipMemcpy(dev_a, a, ROWS * COLUMNS * sizeof(int), hipMemcpyHostToDevice);
    add <<<1, COLUMNS >>> (dev_a, dev_c);

    hipMemcpy(c), dev_c, COLUMNS * sizeof(int), hipMemcpyDeviceToHost);

    for (int y = 0; y < ROWS; y++)              // Output Arrays
    {
        for (int x = 0; x < COLUMNS; x++)
            printf("[%d][%d]=%d ", y, x, c[y][x]);
        printf("\n");
    }
    return 0;
}


#include "hip/hip_runtime.h"
#include "FreeImage.h" // Compile with –lfreeimage  flag
#include "stdio.h"

#define DIM 2000

struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex(float a, float b) : r(a), i(b) {}
    __device__ float magnitude2(void) { return r * r + i * i; }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__ int julia(int x, int y)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);
    int i = 0;
    for (i = 0; i < 200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000) return 0;
    }
    return 1;
}

__global__ void kernel(char* ptr)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * DIM;
    ptr[offset] = julia(x, y);
}

int main()
{
    FreeImage_Initialise();
    atexit(FreeImage_DeInitialise);
    FIBITMAP* bitmap = FreeImage_Allocate(DIM, DIM, 24);

    char charmap[DIM][DIM];
    char* dev_charmap;

    hipMalloc((void**)&dev_charmap, DIM * DIM * sizeof(char));
    dim3 grid(DIM, DIM);
    kernel <<<grid, 1 >>> (dev_charmap);

    hipMemcpy(charmap, dev_charmap, DIM * DIM * sizeof(char), hipMemcpyDeviceToHost);

    RGBQUAD color;
    for (int i = 0; i < DIM; i++) {
        for (int j = 0; j < DIM; j++) {
            color.rgbRed = 0;
            color.rgbGreen = 0;
            color.rgbBlue = 0;
            if (charmap[i][j] == 1)
                color.rgbBlue = 255.0;
            FreeImage_SetPixelColor(bitmap, i, j, &color);
        }
    }

    FreeImage_Save(FIF_PNG, bitmap, "output.png", 0);
    FreeImage_Unload(bitmap);
    hipFree(dev_charmap);

    return 0;
}